
#include <hip/hip_runtime.h>
#include <iostream>

// Kernel definition
__global__ void VecAdd(float* A, float* B, float* C) {
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

int main() {
    // Number of elements
    int N = 512;

    // Allocate host memory
    float* h_A = (float*)malloc(N * sizeof(float));
    float* h_B = (float*)malloc(N * sizeof(float));
    float* h_C = (float*)malloc(N * sizeof(float));

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    // Allocate device memory
    float* d_A;
    float* d_B;
    float* d_C;
    hipMalloc(&d_A, N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_C, N * sizeof(float));

    // Copy host memory to device memory
    hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice);

    // Kernel invocation with N threads
    VecAdd<<<1, N>>>(d_A, d_B, d_C);

    // Copy result from device memory to host memory
    hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    // Print results
    for (int i = 0; i < N; i++) {
        std::cout << h_A[i] << " + " << h_B[i] << " = " << h_C[i] << std::endl;
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}


// #include<iostream>

// __global__ void mykernelfunc(){};

// int main()
// {
//     mykernelfunc<<<1,1>>>();
//     std::cout<<"hello world cuda!\n";
//     return 0;
// }
